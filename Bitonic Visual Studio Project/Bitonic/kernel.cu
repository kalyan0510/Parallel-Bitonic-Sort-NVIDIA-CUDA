#include "hip/hip_runtime.h"
#include <stdio.h>
#include<conio.h>
#include<algorithm>
#include<stdlib.h>
#include "mylib.h"
void sortb(int* a, int size);


__global__ void bitonicSort(int* a, int n, int g, int t) {
	
	int index = blockIdx.x * blockDim.x + threadIdx.x,
		map = (index / (1 << (t - 1)))*(1 << t) + (index % (1 << (t - 1))),
		pos = (map / (1 << g)) % 2,
		m1 = (pos == 0) ? map : (map + (1 << (t - 1))),
		m2=  (pos == 0) ? (map + (1 << (t - 1))):map;
	//  m ap <>  map + 1<<(b-1)
	//printf("%d  %d    index - %d        %d , %d        %d\n", g, b, index, map, map + (1 << (b - 1)), pos);
	atomicMin(&a[m1], atomicMax(&a[m2], a[m1]));
	__syncthreads();
}


double sortb(int* a, int size,int logn2) {
	
	int* array;
	int mem = sizeof(int) * size;
	hipMalloc((void **)&array, sizeof(int)*size);
	hipMemcpy(array, a, sizeof(int) * size, hipMemcpyHostToDevice);

	
	//printf("\nthreads p block = %d\n", size);
	
	int threadsPerBlock = 1024;
	int blocksPerGrid = ((size/2) + threadsPerBlock - 1) / threadsPerBlock;
	clock_t t, t1,td;
	t = clock();
	for (int g = 1; g <= logn2; g++) {
		for (int t = g; t > 0; t--) {
			//printf("g-> %d   t-> %d\n", g, t);
			bitonicSort <<<blocksPerGrid, threadsPerBlock >>>(array, size, g, t);
			//printf("\n\n");
			//
		}
	}
	td = clock();
	hipDeviceSynchronize();
	t1 = clock();
	double time_taken = ((double)(avg2(t1,td)-t)) / CLOCKS_PER_SEC;
	//printf("\n\nfunction exec time: "); printf(" %.3lfs\n\n", time_taken,t,t1);
	//bitonicSort <<<1, (size / 2) >>>(array, size / 2);

	hipMemcpy(a, array, size * sizeof(int), hipMemcpyDeviceToHost);
	//size /= 2;
	//}
	hipFree(array);
	return time_taken;
}
int * intdup(int const * src, size_t len)
{
	int * p = (int *)malloc(len * sizeof(int));
	memcpy(p, src, len * sizeof(int));
	return p;
}
void sortn(int* arr, int n) {
	int * dup = intdup(arr, n);
	clock_t t;
	t = clock();
	std::sort(dup, dup + n);
	t = clock() - t;
	double time_taken = ((double)t) / CLOCKS_PER_SEC;
	printf("exec time of Normal sort: "); printf(" %.3lfs       \nDuplicate array using sequential is sorted %s\n", time_taken, std::is_sorted(dup, dup + n) == 1 ? "YES" : "NO");
}
void wop(int * array, int sz) {
	FILE *f1;
	f1 = fopen("op.txt", "w");
	fprintf(f1, "%d ", sz);
	for (int i = 0; i<sz; i++) {

		fprintf(f1, "%d ", array[i]);
	}
	fclose(f1);

}
double avgfornP(int *arr,int n) {
	double x = 1e-9, itr=1;
	for (int i = 0; i < itr; i++) {
		x += sortb(arr, n, (int)(log(n) / log(2)));
	}
	x = x / itr;																																						
	return x;
}
double avgfornS(int *arr, int n) {
	double x = 0, itr = 1;
	for (int i = 0; i < itr; i++) {
		clock_t t;
		t = clock();
		//sortb(arr, n, (int)(log(n) / log(2)));
		std::sort(arr, arr + n);
		t = clock() - t;
		double time_taken = ((double)t) / CLOCKS_PER_SEC;
		x += time_taken;
	}
	x = x / itr;
	return x;
}
int getAnalysis(int *arr,int size,int w) {
	FILE *f1; double a, b;
	int * dup = intdup(arr,size);
	f1 = fopen("analysis.txt", "a");
	fprintf(f1, "\nAnalysis Report:    \n\n",w);
	fprintf(f1, "   N       time over        time over        is sorted?    speedup:\n");
	fprintf(f1, "   N      serial code      Bitonic CUDA code\n");
	printf( "   N       time over        time over       is sorted?    speedup:\n");
	printf( "   N      serial code      Bitonic CUDA code\n");
	for (int i = 2; i <= 20; i++) {
		memcpy(arr, dup, (1 << i) * sizeof(int));
		a = avgfornS(arr, (1 << i));
		memcpy(arr, dup, (1<<i) * sizeof(int));
		b = avgfornP(arr, (1 << i));
		
		fprintf(f1, "%7d      %.5lf          %.5lf",(1<<i), a, b);
		printf("%7d      %.5lf          %.5lf", (1 << i),a,b);
		fprintf(f1,"          %s     %f\n", std::is_sorted(arr, arr + (1 << i)) == 1 ? "YES" : "NO",a/b );
		printf("          %s     %f\n", std::is_sorted(arr, arr + (1<<i)) == 1 ? "YES" : "NO",  a/b);
		
	}
	fclose(f1);
	return 0;
}
int main(int argc, char **argv) {
	
	int* arr;

	
	int n, s;
	FILE *f = fopen("z.txt", "r");

	if (f == NULL) {
		fprintf(stderr, "File not found.\n");
		return 1;
	}

	
	fscanf(f, "%d", &n);
	printf("size n = %d log n = %d\n", n, (int)(log(n) / log(2)));

	arr = (int*)malloc(n * sizeof(int));
	for (int i = 0; i < n; i++) {
		fscanf(f, "%d", (arr + i));
		//printf(" %d ", arr[i]);
	}
	fclose(f);		
	
	for(int i=0;i<1;i++)
		getAnalysis(arr, n, 1);
	///printf("input is sorted: %s\n\n\n", std::is_sorted(arr, arr + n) == 1 ? "YES" : "NO");
	//----sortn(arr, n);
	clock_t t;
	
	//----double time_taken =sortb(arr, n, (int)(log(n) / log(2)));
	//std::sort(arr, arr + n);
	
	
	
	//---printf("\n");
	//----wop(arr,n);
	/*for (int i = 0; i < n; i++) {
		printf(" %d ", arr[i]);
	}*/
	
	//----printf("\narray using Parallel Sort is sorted: %s\n", std::is_sorted(arr, arr + n)==1?"YES":"NOOOOOOOOOOOOOOOOOOOOOOOOO");
	//-----printf("exec time: "); printf(" %lfs \n\n",time_taken); 
	puts("...");
	getch();
}